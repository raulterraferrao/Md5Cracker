#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include "md5.cu"

#define THREADS_PER_BLOCK 128
#define BLOCKS 1024
#define MAX_STR_LENGTH 16
#define TAM_HASH 33
#define TAM_ALFABETO 26
#define MAX_HASH_PER_KERNEL 8192
#define MIN_HASH_PER_KERNEL 256



#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


struct DeviceReturnStruct{

	int a;

};

__device__ uint d_unhex(unsigned char x)
{
    if(x <= 'F' && x >= 'A')
    {
        return  (uint)(x - 'A' + 10);
    }
    else if(x <= 'f' && x >= 'a')
    {
        return (uint)(x - 'a' + 10);
    }
    else if(x <= '9' && x >= '0')
    {
        return (uint)(x - '0');
    }
    return 0;
}

__device__ void d_md5_to_ints(unsigned char* md5, uint *r0, uint *r1, uint *r2, uint *r3)
{
    uint v0 = 0, v1 = 0, v2 = 0, v3 = 0;
    int i = 0;
    for(i = 0; i < 32; i+=2)
    {
        uint first = d_unhex(md5[i]);
        uint second = d_unhex(md5[i+1]);
        uint both = first * 16 + second;
        both = both << 24;
        if(i < 8)
        {
            v0 = (v0 >> 8 ) | both;
        }
        else if (i < 16)
        {
            v1 = (v1 >> 8) | both;
        }
        else if (i < 24)
        {
            v2 = (v2 >> 8) | both;
        }
        else if(i < 32)
        {
            v3 = (v3 >> 8) | both;
        }
    }

    *r0 = v0;
    *r1 = v1;
    *r2 = v2;
    *r3 = v3;
}

__device__ inline int my_strlen(unsigned char* str){
	int i = 0;
	while(str[i++] != '\0');
	return --i;
}


__device__ void my_strcpy(unsigned char *dest, const unsigned char *src){
  int i = 0;
  do {
    dest[i] = src[i];}
  while (src[i++] != '\0');
}

__device__ void converter(ulong numeroEntrada, unsigned char str[MAX_STR_LENGTH])
{
	int i = 0;  // To store current index in str which is result

	while (numeroEntrada>0)
	{
		// Find remainder
		ulong rem = numeroEntrada%26;

		// If remainder is 0, then a 'Z' must be there in output
		if (rem==0)
		{
			str[i++] = 'z';
			numeroEntrada = (numeroEntrada/26)-1;
		}
		else // If remainder is non-zero
		{
			str[i++] = (rem-1) + 'a';
			numeroEntrada = numeroEntrada/26;
		}
	}
	str[i] = '\0';



}

__global__ void crack(unsigned char *password, ulong* starting_number , uint* d_current_hash_per_kernel ,volatile int* flag ,uint* d_v1, uint* d_v2, uint* d_v3, uint* d_v4 , DeviceReturnStruct *device_return)
{
	const ulong thread_per_block = THREADS_PER_BLOCK;
	const ulong blocks = BLOCKS;
	const ulong step = thread_per_block * blocks;

	const uint v1 = *d_v1;
	const uint v2 = *d_v2;
	const uint v3 = *d_v3;
	const uint v4 = *d_v4;

	const uint current_hash_per_kernel = *d_current_hash_per_kernel;

	unsigned char palavra[MAX_STR_LENGTH] = "";

	int count = 0;
	int totalLen;
	uint c1 = 0, c2 = 0, c3 = 0, c4 = 0;
	ulong len;

	ulong blockIdxx = blockIdx.x;
	ulong blockDimx = blockDim.x;
	ulong threadIdxx = threadIdx.x;


	ulong idx = (blockIdxx*blockDimx + threadIdxx) + *starting_number;

	while(*flag != 1 && count++ < current_hash_per_kernel ){
		totalLen = 1;
		len = idx /(TAM_ALFABETO + 1);
		while(len > 0){
			len /= (TAM_ALFABETO + 1);
			totalLen++;
		}

		converter(idx, palavra);
		md5_vfy(palavra,totalLen, &c1, &c2, &c3, &c4);

		if(c1 == v1 && c2 == v2 && c3 == v3 && c4 == v4)
		{
			my_strcpy(password,palavra);
			*flag = 1;
		}

		idx += step;

	}

}

void h_converter(ulong numeroEntrada, unsigned char pointerPalavra[][255])
{
	unsigned char* str = *pointerPalavra;  // To store result (Excel column name)
	int i = 0;  // To store current index in str which is result

	while (numeroEntrada>0)
	{
		// Find remainder
		ulong rem = numeroEntrada%26;

		// If remainder is 0, then a 'Z' must be there in output
		if (rem==0)
		{
			str[i++] = 'z';
			numeroEntrada = (numeroEntrada/26)-1;
		}
		else // If remainder is non-zero
		{
			str[i++] = (rem-1) + 'a';
			numeroEntrada = numeroEntrada/26;
		}
	}
	str[i] = '\0';

}

int main(int argc,  char *argv[]){

	unsigned char *d_password;
	ulong* d_starting_number;
	ulong h_starting_number = 0;
	int *d_flag;
	int h_flag = 0;
	unsigned char h_password[MAX_STR_LENGTH] = "";

	uint v1,v2,v3,v4;
	uint *d_v1, *d_v2, *d_v3, *d_v4;
	DeviceReturnStruct h_device_return, *d_device_return;


	uint h_current_hash_per_kernel = MIN_HASH_PER_KERNEL;
	uint* d_current_hash_per_kernel;


	//divido o hash em 4 partes
	md5_to_ints((unsigned char*)argv[1],&v1,&v2,&v3,&v4);

	//Saida de erro caso não tiver um hash como entrada no argumento
    if ( argc != 2 )
    {
        fprintf( stderr, "Erro na entrada de argumentos %s \n", argv[0] );
        exit( 1 );
    }

    //Copia o argumento para a variavel hash_entrada
    //memcpy(hash_entrada,argv[1], TAM_HASH);


	hipMalloc( (void**)&d_password, MAX_STR_LENGTH*sizeof(unsigned char));
	cudaCheckErrors("d_password");
	hipMalloc( (void**)&d_flag, sizeof(int));
	cudaCheckErrors("d_flag");
	hipMalloc( (void**)&d_starting_number, sizeof(ulong));
	cudaCheckErrors("d_starting_number");
	hipMalloc( (void**)&d_v1, sizeof(uint));
	cudaCheckErrors("d_v1");
	hipMalloc( (void**)&d_v2, sizeof(uint));
	cudaCheckErrors("d_v2");
	hipMalloc( (void**)&d_v3, sizeof(uint));
	cudaCheckErrors("d_v3");
	hipMalloc( (void**)&d_v4, sizeof(uint));
	cudaCheckErrors("d_v4");
	hipMalloc( (void**)&d_current_hash_per_kernel, sizeof(uint));
	cudaCheckErrors("d_current_hash_per_kernel");

	hipMalloc( (void**)&d_current_hash_per_kernel, sizeof(uint));
	hipMalloc( (void**)&d_device_return, sizeof(DeviceReturnStruct));


	if(d_password ==0  || d_flag ==0)
	{
      printf("couldn't allocate memory\n");
      return 1;
	}

	hipMemcpy( d_flag, &h_flag, sizeof(int),  hipMemcpyHostToDevice);
	cudaCheckErrors("hipMemcpy( d_flag, &h_flag,sizeof(int),  hipMemcpyHostToDevice);");
	hipMemcpy( d_starting_number, &h_starting_number, sizeof(ulong), hipMemcpyHostToDevice);
	cudaCheckErrors("hipMemset( d_starting_number, 0,sizeof(int) );");
	hipDeviceSynchronize();
	cudaCheckErrors("hipDeviceSynchronize();");

	hipMemcpy( d_v1, &v1, sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy( d_v2, &v2, sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy( d_v3, &v3, sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy( d_v4, &v4, sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy( d_password, h_password, MAX_STR_LENGTH*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy( d_current_hash_per_kernel, &h_current_hash_per_kernel, sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy( d_device_return, &h_device_return, sizeof(DeviceReturnStruct), hipMemcpyHostToDevice);


	//run the kernel
	dim3 dimGrid(BLOCKS);
	dim3 dimBlock(THREADS_PER_BLOCK);
	ulong blocks = BLOCKS;
	ulong threads_per_block = THREADS_PER_BLOCK;
	ulong currentNumber = 0;

	int i = 0, count = 0;
	unsigned char palavra[255] = "";
	while(h_flag != 1){
		crack<<<dimGrid, dimBlock>>>(d_password, d_starting_number , d_current_hash_per_kernel ,d_flag, d_v1, d_v2, d_v3, d_v4, d_device_return);
		cudaCheckErrors("crack");
		//cudaCheckErrors("hipDeviceSynchronize();");
		hipMemcpy( &h_flag, d_flag,sizeof(int), hipMemcpyDeviceToHost );
		//cudaCheckErrors("hipMemcpy( &h_flag, d_flag,sizeof(int), hipMemcpyDeviceToHost )");


		//cudaCheckErrors("hipMemcpy( d_starting_number, &h_starting_number, sizeof(ulong), hipMemcpyHostToDevice );");
		currentNumber = h_starting_number;
		h_starting_number += threads_per_block*blocks*h_current_hash_per_kernel;
		h_current_hash_per_kernel = h_current_hash_per_kernel + MIN_HASH_PER_KERNEL;
		if(h_current_hash_per_kernel  > MAX_HASH_PER_KERNEL){
			h_current_hash_per_kernel = MAX_HASH_PER_KERNEL;
		}
		h_converter(currentNumber, &palavra);

		if(count++%1 == 0){
			printf("i = %d, number = %lu, palavra = %s\n",i, currentNumber, palavra);
			printf("hash_entrada = %s\n",argv[1]);
		}

		hipMemcpy( d_current_hash_per_kernel, &h_current_hash_per_kernel, sizeof(uint), hipMemcpyHostToDevice );
		hipMemcpy( d_starting_number, &h_starting_number, sizeof(ulong), hipMemcpyHostToDevice );

		i++;
	}
	//cudaCheckErrors("hipMemcpy( hash_entrada, d_hash_entrada,num_bytes, hipMemcpyDeviceToHost );");
	hipMemcpy( h_password, d_password, MAX_STR_LENGTH, hipMemcpyDeviceToHost );
	//cudaCheckErrors("hipMemcpy( h_password, d_password,num_bytes*2, hipMemcpyDeviceToHost );");


	printf("---------------------------------------\n");
	printf("Password = %s\n",h_password);
	printf("Flag %d \n", h_flag );


	 hipFree( d_password);
	 hipFree( d_flag);
	 hipFree( d_starting_number);
	 hipFree( d_v1);
	 hipFree( d_v2);
	 hipFree( d_v3);
	 hipFree( d_v4);
	 hipFree( d_device_return);


	return 0;
}

