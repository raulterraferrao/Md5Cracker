#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include "md5.cu"

#define THREADS_PER_BLOCK 128
#define BLOCKS 1024
#define MAX_STR_LENGTH 16
#define TAM_HASH 33
#define TAM_ALFABETO 26
#define MAX_HASH_PER_KERNEL 8192
#define MIN_HASH_PER_KERNEL 256


#define CONST_CHARSET "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789"
#define CONST_CHARSET_LENGTH (sizeof(CONST_CHARSET) - 1)



#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


struct DeviceReturnStruct{

	int a;

};

__device__ uint d_unhex(unsigned char x)
{
    if(x <= 'F' && x >= 'A')
    {
        return  (uint)(x - 'A' + 10);
    }
    else if(x <= 'f' && x >= 'a')
    {
        return (uint)(x - 'a' + 10);
    }
    else if(x <= '9' && x >= '0')
    {
        return (uint)(x - '0');
    }
    return 0;
}

__device__ void d_md5_to_ints(unsigned char* md5, uint *r0, uint *r1, uint *r2, uint *r3)
{
    uint v0 = 0, v1 = 0, v2 = 0, v3 = 0;
    int i = 0;
    for(i = 0; i < 32; i+=2)
    {
        uint first = d_unhex(md5[i]);
        uint second = d_unhex(md5[i+1]);
        uint both = first * 16 + second;
        both = both << 24;
        if(i < 8)
        {
            v0 = (v0 >> 8 ) | both;
        }
        else if (i < 16)
        {
            v1 = (v1 >> 8) | both;
        }
        else if (i < 24)
        {
            v2 = (v2 >> 8) | both;
        }
        else if(i < 32)
        {
            v3 = (v3 >> 8) | both;
        }
    }

    *r0 = v0;
    *r1 = v1;
    *r2 = v2;
    *r3 = v3;
}

__device__ inline int my_strlen(char* str){
	int i = 0;
	while(str[i++] != '\0');
	return --i;
}

int host_my_strlen(char* str){
    int i = 0;
    while(str[i++] != '\0');
    return --i;
}


__device__ void my_strcpy(unsigned char *dest, const unsigned char *src){
  int i = 0;
  do {
    dest[i] = src[i];}
  while (src[i++] != '\0');
}


//This function transform the respective number that is passed as numeroEntrada to a string with the chars of charset

__device__ void converter(ulong numeroEntrada, unsigned char str[MAX_STR_LENGTH],char* charset)
{
            int size;
            size = my_strlen(charset);


            int i = 0;  // To store current index in str which is result

	while (numeroEntrada>0)
	{
		// Find remainder
		ulong rem = numeroEntrada%size;

		// If remainder is 0, then a 'Z' must be there in output
		if (rem==0)
		{
			str[i++] = charset[size-1];
			numeroEntrada = (numeroEntrada/size)-1;
		}
		else // If remainder is non-zero
		{
			str[i++] = charset[(rem-1)];
			numeroEntrada = numeroEntrada/size;
		}
	}
	str[i] = '\0';



}

__global__ void crack(unsigned char *password, ulong* starting_number , uint* d_current_hash_per_kernel ,volatile int* flag ,uint* d_v1, uint* d_v2, uint* d_v3, uint* d_v4 ,char* charset, DeviceReturnStruct *device_return)
{
	const ulong thread_per_block = THREADS_PER_BLOCK;
	const ulong blocks = BLOCKS;
	const ulong step = thread_per_block * blocks;

	const uint v1 = *d_v1;
	const uint v2 = *d_v2;
	const uint v3 = *d_v3;
	const uint v4 = *d_v4;

            int size;
            size = my_strlen(charset);

	const uint current_hash_per_kernel = *d_current_hash_per_kernel;

	unsigned char palavra[MAX_STR_LENGTH] = "";

	int count = 0;
	int totalLen;
	uint c1 = 0, c2 = 0, c3 = 0, c4 = 0;
	ulong len;

	ulong blockIdxx = blockIdx.x;
	ulong blockDimx = blockDim.x;
	ulong threadIdxx = threadIdx.x;


	ulong idx = (blockIdxx*blockDimx + threadIdxx) + *starting_number;

	while(*flag != 1 && count++ < current_hash_per_kernel ){
		totalLen = 1;
		len = idx /(size + 1);
		while(len > 0){
			len /= (size + 1);
			totalLen++;
		}

		converter(idx, palavra,charset);
		md5_vfy(palavra,totalLen, &c1, &c2, &c3, &c4);

		if(c1 == v1 && c2 == v2 && c3 == v3 && c4 == v4)
		{
			my_strcpy(password,palavra);
			*flag = 1;
		}

		idx += step;

	}

}

void h_converter(ulong numeroEntrada, unsigned char pointerPalavra[][255],char* charset)
{
	int size;
             size = host_my_strlen(charset);

            unsigned char* str = *pointerPalavra;  // To store result (Excel column name)
	int i = 0;  // To store current index in str which is result

	while (numeroEntrada>0)
	{
		// Find remainder
		ulong rem = numeroEntrada%size;

		// If remainder is 0, then a 'Z' must be there in output
		if (rem==0)
		{
			str[i++] = charset[size-1];
			numeroEntrada = (numeroEntrada/size)-1;
		}
		else // If remainder is non-zero
		{
			str[i++] = charset[(rem-1)];
			numeroEntrada = numeroEntrada/size;
		}
	}
	str[i] = '\0';

}

void n_converter(ulong numeroEntrada, unsigned char pointerPalavra[][255],char* charset)
{
    int size;
    size = host_my_strlen(charset);
    printf("%d",size);

    unsigned char* str = *pointerPalavra;  // To store result (Excel column name)
    int i = 0;  // To store current index in str which is result

    while (numeroEntrada>0)
    {
        // Find remainder
        ulong rem = numeroEntrada%size;

        // If remainder is 0, then a 'Z' must be there in output
        if (rem==0)
        {
            str[i++] = charset[size-1];
            numeroEntrada = (numeroEntrada/size)-1;
        }
        else // If remainder is non-zero
        {
            str[i++] = charset[(rem-1)];
            numeroEntrada = numeroEntrada/size;
        }
    }
    str[i] = '\0';

}

int main(int argc,  char *argv[]){

	/*======================================
                RESPECTIVE NUMBERS OF CHARSET
            ========================================

            0: a-z
            1: A-Z
            2: 0-9
            3: a-z A-Z
            4: a-z 0-9
            5: A-Z 0-9
            6: a-z A-z 0-9

            */
            char *charset,*d_charset;
            int charset_choice,charset_flag = 0,hash_flag = 0;
            unsigned char hash_entrada[TAM_HASH];
            //unsigned char teste[255] = "";
            //ulong testenumero;

            unsigned char *d_password;
	ulong* d_starting_number;
	ulong h_starting_number = 0;
	int *d_flag;
	int h_flag = 0;
	unsigned char h_password[MAX_STR_LENGTH] = "";

	uint v1,v2,v3,v4;
	uint *d_v1, *d_v2, *d_v3, *d_v4;
	DeviceReturnStruct h_device_return, *d_device_return;


	uint h_current_hash_per_kernel = MIN_HASH_PER_KERNEL;
	uint* d_current_hash_per_kernel;

             printf("\n\nCuda Md5 Brute Force Cracker - made by: Raul Terra Ferrão & Victor Terra Ferrão\n\n");

              do{
                     printf("------- Please paste the md5 hash below -------\n\n");
                     scanf("%s",hash_entrada);
                     (host_my_strlen((char*)hash_entrada) != 32 ) ? (printf("\n\nYour md5 hash is wrong, it must be 32 char length \n\n")) : (hash_flag=1);
            }while(hash_flag == 0);

             printf("\nThe hash is : %s\n\n",hash_entrada);

             do{
                     printf("------- Please choose the number of the charset that you want -------\n");
                     printf("0: a-z\n");
                     printf("1: A-Z\n");
                     printf("2: 0-9\n");
                     printf("3: a-z A-Z\n");
                     printf("4: a-z 0-9\n");
                     printf("5: A-Z 0-9\n");
                     printf("6: a-z A-z 0-9\n");
                     printf("----------------------------------------------------------------------\n\n");
                     scanf("%d",&charset_choice);
                     (charset_choice > 6 ||  charset_choice < 0 ) ? (printf("\n\nYou need to write a number between 0 and 6\n\n")) : (charset_flag=1);
             }while(charset_flag == 0);

             printf("The number of charset is : %d\n\n",charset_choice);

             switch(charset_choice){

                case 0:
                      charset =  (char*) malloc (sizeof (char) * 26);
                      strcpy (charset,"abcdefghijklmnopqrstuvwxyz");
                      hipMalloc( (void**)&d_charset, sizeof(char) * 26);
                      cudaCheckErrors("d_charset");
                      hipMemcpy( d_charset, charset, sizeof(char) * 26, hipMemcpyHostToDevice);
                      printf("%s\n",charset);
                break;
                case 1:
                      charset =  (char*) malloc (sizeof (char) * 26);
                      strcpy (charset,"ABCDEFGHIJKLMNOPQRSTUVWXYZ");
                      hipMalloc( (void**)&d_charset, sizeof(char) * 26);
                      cudaCheckErrors("d_charset");
                      hipMemcpy( d_charset, charset, sizeof(char) * 26, hipMemcpyHostToDevice);
                      printf("%s\n",charset);
                break;
                case 2:
                      charset =  (char*) malloc (sizeof (char) * 10);
                      strcpy (charset,"0123456789");
                      hipMalloc( (void**)&d_charset, sizeof(char) * 10);
                      cudaCheckErrors("d_charset");
                      hipMemcpy( d_charset, charset, sizeof(char) * 10, hipMemcpyHostToDevice);
                      printf("%s\n",charset);
                break;
                case 3:
                      charset =  (char*) malloc (sizeof (char) * 52);
                      strcpy (charset,"abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ");
                      hipMalloc( (void**)&d_charset, sizeof(char) * 52);
                      cudaCheckErrors("d_charset");
                      hipMemcpy( d_charset, charset, sizeof(char) * 52, hipMemcpyHostToDevice);
                      printf("%s\n",charset);
                break;
                case 4:
                      charset =  (char*) malloc (sizeof (char) * 36);
                      strcpy (charset,"abcdefghijklmnopqrstuvwxyz0123456789");
                      hipMalloc( (void**)&d_charset, sizeof(char) * 36);
                      cudaCheckErrors("d_charset");
                      hipMemcpy( d_charset, charset, sizeof(char) * 36, hipMemcpyHostToDevice);
                      printf("%s\n",charset);
                break;
                case 5:
                      charset =  (char*) malloc (sizeof (char) * 36);
                      strcpy (charset,"ABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789");
                      hipMalloc( (void**)&d_charset, sizeof(char) * 36);
                      cudaCheckErrors("d_charset");
                      hipMemcpy( d_charset, charset, sizeof(char) * 36, hipMemcpyHostToDevice);
                      printf("%s\n",charset);
                break;
                case 6:
                      charset =  (char*) malloc (sizeof (char) * 62);
                      strcpy (charset,"abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789");
                      hipMalloc( (void**)&d_charset, sizeof(char) * 62);
                      cudaCheckErrors("d_charset");
                      hipMemcpy( d_charset, charset, sizeof(char) * 62, hipMemcpyHostToDevice);
                      printf("%s\n",charset);
                break;
             }

            md5_to_ints((unsigned char*)hash_entrada,&v1,&v2,&v3,&v4);
            //printf("v1,v2,v3,v4 %u,%u,%u,%u\n",v1,v2,v3,v4);



	hipMalloc( (void**)&d_password, MAX_STR_LENGTH*sizeof(unsigned char));
	cudaCheckErrors("d_password");
	hipMalloc( (void**)&d_flag, sizeof(int));
	cudaCheckErrors("d_flag");
	hipMalloc( (void**)&d_starting_number, sizeof(ulong));
	cudaCheckErrors("d_starting_number");
	hipMalloc( (void**)&d_v1, sizeof(uint));
	cudaCheckErrors("d_v1");
	hipMalloc( (void**)&d_v2, sizeof(uint));
	cudaCheckErrors("d_v2");
	hipMalloc( (void**)&d_v3, sizeof(uint));
	cudaCheckErrors("d_v3");
	hipMalloc( (void**)&d_v4, sizeof(uint));
	cudaCheckErrors("d_v4");
	hipMalloc( (void**)&d_current_hash_per_kernel, sizeof(uint));
	cudaCheckErrors("d_current_hash_per_kernel");



	hipMalloc( (void**)&d_current_hash_per_kernel, sizeof(uint));
	hipMalloc( (void**)&d_device_return, sizeof(DeviceReturnStruct));


	if(d_password ==0  || d_flag ==0)
	{
      printf("couldn't allocate memory\n");
      return 1;
	}

	hipMemcpy( d_flag, &h_flag, sizeof(int),  hipMemcpyHostToDevice);
	cudaCheckErrors("hipMemcpy( d_flag, &h_flag,sizeof(int),  hipMemcpyHostToDevice);");
	hipMemcpy( d_starting_number, &h_starting_number, sizeof(ulong), hipMemcpyHostToDevice);
	cudaCheckErrors("hipMemset( d_starting_number, 0,sizeof(int) );");
	hipDeviceSynchronize();
	cudaCheckErrors("hipDeviceSynchronize();");

	hipMemcpy( d_v1, &v1, sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy( d_v2, &v2, sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy( d_v3, &v3, sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy( d_v4, &v4, sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy( d_password, h_password, MAX_STR_LENGTH*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy( d_current_hash_per_kernel, &h_current_hash_per_kernel, sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy( d_device_return, &h_device_return, sizeof(DeviceReturnStruct), hipMemcpyHostToDevice);


	//run the kernel
	dim3 dimGrid(BLOCKS);
	dim3 dimBlock(THREADS_PER_BLOCK);
	ulong blocks = BLOCKS;
	ulong threads_per_block = THREADS_PER_BLOCK;
	ulong currentNumber = 0;

	int i = 0, count = 0;
	unsigned char palavra[255] = "";
	while(h_flag != 1){
		crack<<<dimGrid, dimBlock>>>(d_password, d_starting_number , d_current_hash_per_kernel ,d_flag, d_v1, d_v2, d_v3, d_v4,d_charset, d_device_return);
		cudaCheckErrors("crack");
		//cudaCheckErrors("hipDeviceSynchronize();");
		hipMemcpy( &h_flag, d_flag,sizeof(int), hipMemcpyDeviceToHost );
		//cudaCheckErrors("hipMemcpy( &h_flag, d_flag,sizeof(int), hipMemcpyDeviceToHost )");


		//cudaCheckErrors("hipMemcpy( d_starting_number, &h_starting_number, sizeof(ulong), hipMemcpyHostToDevice );");
		currentNumber = h_starting_number;
		h_starting_number += threads_per_block*blocks*h_current_hash_per_kernel;
		h_current_hash_per_kernel = h_current_hash_per_kernel + MIN_HASH_PER_KERNEL;
		if(h_current_hash_per_kernel  > MAX_HASH_PER_KERNEL){
			h_current_hash_per_kernel = MAX_HASH_PER_KERNEL;
		}
		h_converter(currentNumber, &palavra,charset);

		if(count++%1 == 0){
			printf("i = %d, number = %lu, palavra = %s\n",i, currentNumber, palavra);
			//printf("hash_entrada = %s\n",argv[1]);
		}

		hipMemcpy( d_current_hash_per_kernel, &h_current_hash_per_kernel, sizeof(uint), hipMemcpyHostToDevice );
		hipMemcpy( d_starting_number, &h_starting_number, sizeof(ulong), hipMemcpyHostToDevice );

		i++;
	}
	//cudaCheckErrors("hipMemcpy( hash_entrada, d_hash_entrada,num_bytes, hipMemcpyDeviceToHost );");
	hipMemcpy( h_password, d_password, MAX_STR_LENGTH, hipMemcpyDeviceToHost );
	//cudaCheckErrors("hipMemcpy( h_password, d_password,num_bytes*2, hipMemcpyDeviceToHost );");


	printf("---------------------------------------\n");
	printf("Password = %s\n",h_password);
	printf("Flag %d \n", h_flag );


	 hipFree( d_password);
	 hipFree( d_flag);
	 hipFree( d_starting_number);
	 hipFree( d_v1);
	 hipFree( d_v2);
	 hipFree( d_v3);
	 hipFree( d_v4);
	 hipFree( d_device_return);


	return 0;
}

